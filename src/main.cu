//positionFile format
//diam1 diam2...
//
//t K U p1_x p1_y p1_vx p1_vy p2_x...
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <sstream>

#define NT 1024
#define NB 512

typedef unsigned int uint;

template <typename T>
T pow_tmp(T x, int y){
	T xx = (T)1;
	if (y > 0){
		for (int i = 1; i <= y; i++) {
			xx *= x;
		}
	}
	else{
		for (int i = -1; i >= y; i--) {
			xx /= x;
		}
	}

	return xx;
}

//---settings---//
const uint D = 2;
uint ID;
uint NP;
double T;
double tmax;
//--------------//

const double pi = 4 * atan(1.0);
__global__ void xx0t(double *xx0t, double *x, double L, uint l, uint d){
    uint i_block = blockIdx.x;
    uint i_local = threadIdx.x;
    uint i_global = i_block * blockDim.x + i_local;
    double x12;
    for(uint i = i_global; i < l; i += NB*NT){
        x12 = x[i]-x[i%d];
        if(x12 < -0.5 * L) x12 += L;
        if(x12 > 0.5 * L) x12 -= L;
            
        xx0t[i] = x12 * x12;
    }
}
__global__ void reductionMsd(double *out, double *xx0t, uint l){
    uint i_block = blockIdx.x;
    uint i_local = threadIdx.x;
    uint i_global = i_block * blockDim.x + i_local;

    __shared__ double f[NT];

    uint remain, reduce;
    uint ib = i_block;
    for(uint i = i_global; i < l; i += NB*NT){
        f[i_local] = xx0t[i];
        __syncthreads();

        for(uint j = NT; j > 1; j = remain){
            reduce = j >> 1;
            remain = j - reduce;
            if((i_local < reduce) && (i + remain < l)){
                f[i_local] += f[i_local+remain];
            }
            __syncthreads();
        }
        if(i_local == 0){
            out[ib] = f[0];
        }
        __syncthreads();
        ib += NB;
    }
}

int main(int argc, char** argv){
    ID = atoi(argv[1]);
    NP = atoi(argv[2]);
	T = atof(argv[3]);
    uint timescale= atoi(argv[4]);
    tmax = pow_tmp(2., timescale);
    
    std::cout << "---settings---" << std::endl;
    std::cout << "ID: [1, " << ID << "]" << std::endl;
    std::cout << "D: " << D << std::endl;
    std::cout << "NP: " << NP << std::endl;
    std::cout << "T: " << T << std::endl;
    std::cout << "timescale: " << timescale << std::endl;
    std::cout << "--------------" << std::endl << std::endl;

    //Variables
    double *diam;
    double *x, *x_dev;
    double *xx0t_dev[2];
    double *t, *dt;
    diam = new double[NP];

    const double a0 = 1.;
    const double a1 = a0 * 1;
    const double a2 = a0 * 1.4;
    double dnsty = 0.8;
    double L = sqrt((double)NP/dnsty);

    uint Nt;

    //find dt, diam
    std::ostringstream positionName0;
	positionName0 << "../../pos/N"<< argv[2] << "/T" << argv[3] << "/posBD_N" << argv[2] << "_T" << argv[3] << "_id1.data";
	std::ifstream positionFile;
	positionFile.open(positionName0.str().c_str());
	std::cout << "Loading " << positionName0.str() << " for find dt, diam" << std::endl;
	double t1, t2, DA, DK, DU, DX, DV;
	for (uint n = 0; n < NP; n++) {
		positionFile >> diam[n];
	}
	positionFile >> t1 >> DK >> DU;
	for (int n = 0; n < NP; n++) {
        positionFile >> DX >> DX >> DV >> DV;
	}
	positionFile >> t2;
    positionFile.close();
    std::cout << "dt = " << t2-t1 << std::endl;
    Nt = 0;
	double ttmp = 10 * (t2-t1);
	while (ttmp < tmax) {
		ttmp *= 1.1;
		Nt++;
    }

    //newMemory
    x = new double[ID*Nt*NP*D];
    t = new double[Nt];
	dt = new double[Nt - 1];
    hipMalloc((void**)&x_dev, ID * Nt * NP * D * sizeof(double));
    hipMalloc((void**)&xx0t_dev[0], ID * Nt * NP * D * sizeof(double));
    hipMalloc((void**)&xx0t_dev[1], ID * Nt * NP * D * sizeof(double));

    //loadFile
    for (short i = 0; i < ID; i++){
        // positionFile: t pi_x pi_y...
        std::ostringstream positionName;
        positionName << "../../pos/N"<< argv[2] << "/T" << argv[3];
        positionName << "/posBD_N" << argv[2] << "_T" << argv[3] << "_id" << i+1 << ".data";
	    positionFile.open(positionName.str().c_str());
		std::cout << "Loading " << positionName.str() << "..." << std::endl;
    	for (int n = 0; n < NP; n++) {
	    	positionFile >> DA;
    	}
		for (int nt = 0; nt < Nt; nt++){
    			positionFile >> t[nt] >> DK >> DU;
			for (int n = 0; n < NP; n++){
                positionFile >> x[nt*ID*NP*D + i*NP*D + n*D];
                positionFile >> x[nt*ID*NP*D + i*NP*D + n*D + 1];
                positionFile >> DV >> DV;
			}
		}
		positionFile.close();
		std::cout << " -> done" << std::endl;
    }
    hipMemcpy(x_dev, x, Nt*ID*NP*D * sizeof(double), hipMemcpyHostToDevice);

	for (short nt = 1; nt <= Nt - 1; nt++){
		dt[nt - 1] = t[nt] - t[0];
    }
    
    //analise
    std::cout << "Recording msd_N" << argv[2] << "_T_" << argv[3] << "..." << std::endl;
    std::ostringstream msdName;
	msdName << "./data/msd_N" << argv[2] << "_T" << argv[3] << ".data";
	std::ofstream msdFile;
    msdFile.open(msdName.str().c_str());
    double *msd;
    msd = new double[Nt-1];

    xx0t<<<NB,NT>>>(xx0t_dev[0], x_dev, L, Nt*ID*NP*D, ID*NP*D);
    uint flip;
    for(uint nt = 1; nt <= Nt - 1; nt++){
        flip = 0;
        for(uint l = ID*NP*D; l > 1; l = (l+NT-1)/NT){
            flip = !flip;
            reductionMsd<<<NB,NT>>>(&xx0t_dev[flip][nt*ID*NP*D], &xx0t_dev[!flip][nt*ID*NP*D], l);
        }
        hipMemcpy(&msd[nt - 1], &xx0t_dev[flip][nt*ID*NP*D], sizeof(double), hipMemcpyDeviceToHost);
    }
    for (int nt = 1; nt <= Nt - 1; nt++){
		msdFile << dt[nt - 1] << " " << msd[nt - 1]/(ID*NP) << std::endl;
    }
    delete[] msd;
    msdFile.close();
    
    
    //deleteMemory
    delete[] diam;
    delete[] x;
    delete[] t;
    delete[] dt;
    hipFree(x_dev);
    hipFree(xx0t_dev[0]);
    hipFree(xx0t_dev[1]);
    return 0;
}